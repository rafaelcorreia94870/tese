
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

#pragma hd_warning_disable
template<class Function>
__host__ __device__
void invoke(Function f)
{
  f();
}

struct host_only
{
  __host__
  void operator()()
  {
    std::cout << "host_only()" << std::endl;
  }
};

struct device_only
{
  __device__
  void operator()()
  {
    printf("device_only(): thread %d\n", threadIdx.x);
  }
};

__global__
void kernel()
{
  // use from device with device functor
  invoke(device_only());

  // XXX error
  // invoke(host_only());
}

int main()
{
  // use from host with host functor
  

  kernel<<<1,1>>>();
  hipDeviceSynchronize();

  // XXX error
  // invoke(device_only());

  return 0;
}