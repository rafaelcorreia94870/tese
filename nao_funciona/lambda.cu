#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <list>
#include <array>
#include <map>
#include <deque>
#include <iterator> 
#include <hip/hip_runtime.h>
#include <type_traits>


template <typename T, typename Func>
__global__ void mapKernel(T* d_array, int size, Func func) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        d_array[idx] = func(d_array[idx]);
    }
}
template <typename Iterator, typename Func>
void map(Iterator& container, Func& func) {
    using T = typename Iterator::value_type;
    std::vector<T> temp;  
    
    for (auto it = container.begin(); it != container.end(); ++it) {
        temp.push_back(*it);
    }

    size_t size = temp.size(); 
    T* d_array;
    size_t bytes = size * sizeof(T);
    
    
    hipMalloc(&d_array, bytes);
    hipMemcpy(d_array, temp.data(), bytes, hipMemcpyHostToDevice);

    
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    __device__ auto device_func = [=] __device__ (T x) { return func(x); };
    mapKernel<<<numBlocks, blockSize>>>(d_array, size, device_func);

    
    hipMemcpy(temp.data(), d_array, bytes, hipMemcpyDeviceToHost);
    hipFree(d_array);

    
    std::copy(temp.begin(), temp.end(), container.begin());
}

int increment(int x) {
    return x + 1;
}

int main() {
    std::vector<int> intvec = {0, 1, 2, 3};

    map(intvec, increment);

    std::cout << "\nInt vec\n";
    for (int v : intvec) {
        std::cout << v << " ";
    }

    return 0;
}
