#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <algorithm>
#include <cmath>
#include <list>
#include <hip/hip_runtime.h>


template <typename T, typename Func>
__global__ void mapKernel(T* d_array, int size, Func func) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_array[idx] = func(d_array[idx]);
    }
}


struct IntensiveComputation {
    __device__ float operator()(float x) const {
        for (int i = 0; i < 100; ++i) { 
            x = sin(x) * cos(x) + log(x + 1.0f);
        }
        return x;
    }
};



    float CpuIntensiveComputation(float x) {
        for (int i = 0; i < 100; ++i) { 
            x = sin(x) * cos(x) + log(x + 1.0f);
        }
        return x;
    }



template <typename Iterator, typename Func>
void map(Iterator& container, Func& func) {

    using T = typename Iterator::value_type;
    std::vector<T> temp; 
    if constexpr (!std::is_same_v<Iterator, std::vector<T>>){
        for (auto it = container.begin(); it != container.end(); ++it) {
            temp.push_back(*it);
        }
        size_t size = temp.size(); 
        T* d_array;
        size_t bytes = size * sizeof(T);

        hipMalloc(&d_array, bytes);
        hipMemcpy(d_array, temp.data(), bytes, hipMemcpyHostToDevice);
        int blockSize = 256;
        int numBlocks = (size + blockSize - 1) / blockSize;
        mapKernel<<<numBlocks, blockSize>>>(d_array, size, func);
    
        hipMemcpy(temp.data(), d_array, bytes, hipMemcpyDeviceToHost);
        std::copy(temp.begin(), temp.end(), container.begin());
        hipFree(d_array);

    }
    else{

        size_t size = container.size();
        T* d_array;
        size_t bytes = size * sizeof(T);



        hipMalloc(&d_array, bytes);
        hipMemcpy(d_array, container.data(), bytes, hipMemcpyHostToDevice);

        int blockSize = 256;
        int numBlocks = (size + blockSize - 1) / blockSize;

        mapKernel<<<numBlocks, blockSize>>>(d_array, size, func);
        hipDeviceSynchronize();

        hipMemcpy(container.data(), d_array, bytes, hipMemcpyDeviceToHost);
        hipFree(d_array);
    }
}


template <typename Container, typename Func>
void cpuMap(Container& container, Func func) {
    for (auto& elem : container) {
        elem = func(elem);
    }
}

int main() {
    const size_t N = 10000000;  

    //N elementos com 2.0f
    std::vector<float> cudaVec(N, 2.0f);
    std::vector<float> cpuVec(N, 2.0f);
    std::list<float> cudaVecList(N, 2.0f);

    
    auto startCuda = std::chrono::high_resolution_clock::now();
    map(cudaVec, IntensiveComputation());
    auto endCuda = std::chrono::high_resolution_clock::now();
    auto cudaDuration = std::chrono::duration_cast<std::chrono::milliseconds>(endCuda - startCuda);

    auto startCudaList = std::chrono::high_resolution_clock::now();
    map(cudaVecList, IntensiveComputation());
    auto endCudaList = std::chrono::high_resolution_clock::now();
    auto cudaDurationList = std::chrono::duration_cast<std::chrono::milliseconds>(endCudaList - startCudaList);

    
    auto startCpu = std::chrono::high_resolution_clock::now();
    std::transform(cpuVec.begin(),cpuVec.end(),cpuVec.begin(), CpuIntensiveComputation);
    auto endCpu = std::chrono::high_resolution_clock::now();
    auto cpuDuration = std::chrono::duration_cast<std::chrono::milliseconds>(endCpu - startCpu);

    
    bool resultsMatch = true;
    for (size_t i = 0; i < N; ++i) {
        //std::cout << "cudaVec[" << i << "] = " << cudaVec[i] << " cpuVec[" << i << "] = " << cpuVec[i] << "\n";
        if (std::abs(cudaVec[i] - cpuVec[i]) > 1e-6) {
            resultsMatch = false;
            break;
        }
    }

    bool resultsMatchList = true;
    auto cudaIter = cudaVecList.begin();
    for (size_t i = 0; i < N; ++i, ++cudaIter) {
        if (std::abs(*cudaIter - cpuVec[i]) > 1e-6) {
            std::cout << "cudaVecList[" << i << "] = " << *cudaIter << " cpuVec[" << i << "] = " << cpuVec[i] << "\n";
            resultsMatchList = false;
            break;
        }
    }
    
    std::cout << "CUDA Map Time: " << cudaDuration.count() << " ms\n";
    std::cout << "CUDA Map Time List: " << cudaDurationList.count() << " ms\n";
    std::cout << "CPU Map Time: " << cpuDuration.count() << " ms\n";
    std::cout << "Results Match: " << (resultsMatch ? "Yes" : "No") << "\n";
    std::cout << "Results Match List: " << (resultsMatchList ? "Yes" : "No") << "\n";


    return 0;
}
