#include <iostream>
#include <vector>
#include <numeric>
#include <hip/hip_runtime.h>
#include <chrono>
#include "fused_gpu_vec.cuh"  
#include "experimental_fusion.cuh"  
#include "kernel_op.cuh"

#include "../../experimental/includes/framework/rafa.cuh"

std::chrono::duration<double> twointensivecomputations_old(size_t N, size_t loop_count) {
    auto start = std::chrono::high_resolution_clock::now();
    rafa::vector<float> vec(N, 1.0f);
    rafa::vector<float> result(N);

    vec.smart_map(BenchmarkingComputations(loop_count)).smart_map(BenchmarkingComputations(loop_count), result).execute();
    
    auto end = std::chrono::high_resolution_clock::now();
    vec.clear();
    result.clear();

    //result.print();
    hipDeviceReset();


    //std::cout << result[0];
    return end - start;

}

std::chrono::duration<double> tensimplecomputations_old(size_t N) {
    auto start = std::chrono::high_resolution_clock::now();
    rafa::vector<float> vec(N, 1.0f);
    rafa::vector<float> result(N);

    vec.smart_map(SimpleComputation()).smart_map(SimpleComputation()).smart_map(SimpleComputation()).smart_map(SimpleComputation()).smart_map(SimpleComputation()).smart_map(SimpleComputation()).smart_map(SimpleComputation()).smart_map(SimpleComputation()).smart_map(SimpleComputation()).smart_map(SimpleComputation()).execute();
    
    auto end = std::chrono::high_resolution_clock::now();
    vec.clear();
    result.clear();

    hipDeviceReset();

    return end - start;
}

void two_intenvisve_benchmark(size_t N, size_t loop_count_1, size_t loop_count_2, size_t loop_count_3) {
    std::cout << "Iteration,Loop_Count,Expression Time (ns),GPU Vector Time (ns),Old Implementation Time (ns)" << std::endl;
    for (int i = 0; i < 20; i++) {
        auto expr = twointensivecomputations_expr(N, loop_count_1);
        auto gpu_vec = twointensivecomputations_gpu_vec(N, loop_count_1);
        auto old_impl = twointensivecomputations_old(N, loop_count_1);

        std::cout << i + 1 << "," << loop_count_1 << ","
                  << std::chrono::duration_cast<std::chrono::nanoseconds>(expr).count() << ","
                  << std::chrono::duration_cast<std::chrono::nanoseconds>(gpu_vec).count() << ","
                  << std::chrono::duration_cast<std::chrono::nanoseconds>(old_impl).count() << std::endl;


        auto expr2 = twointensivecomputations_expr(N, loop_count_2);
        auto gpu_vec2 = twointensivecomputations_gpu_vec(N, loop_count_2);
        auto old_impl2 = twointensivecomputations_old(N, loop_count_2);
        std::cout << i + 1 << "," << loop_count_2 << ","
                  << std::chrono::duration_cast<std::chrono::nanoseconds>(expr2).count() << ","
                  << std::chrono::duration_cast<std::chrono::nanoseconds>(gpu_vec2).count() << ","
                  << std::chrono::duration_cast<std::chrono::nanoseconds>(old_impl2).count() << std::endl;
        auto expr3 = twointensivecomputations_expr(N, loop_count_3);
        auto gpu_vec3 = twointensivecomputations_gpu_vec(N, loop_count_3);
        auto old_impl3 = twointensivecomputations_old(N, loop_count_3);

        std::cout << i + 1 << "," << loop_count_3 << ","
                  << std::chrono::duration_cast<std::chrono::nanoseconds>(expr3).count() << ","
                  << std::chrono::duration_cast<std::chrono::nanoseconds>(gpu_vec3).count() << ","
                  << std::chrono::duration_cast<std::chrono::nanoseconds>(old_impl3).count() << std::endl;
    }
}

void ten_simple_computations_benchmark(size_t MIN_N, size_t MAX_N) {

    std::cout << "Iteration,N,Expression Time (ns),GPU Vector Time (ns),Old Implementation Time (ns)" << std::endl;
    for (size_t N = MIN_N; N <= MAX_N; N *= 2) {
        for(int i = 0; i < 20; i++) {
            auto expr = tensimplecomputations_expr(N);
            auto gpu_vec = tensimplecomputations_gpu_vec(N);
            auto old_impl = tensimplecomputations_old(N);

            std::cout << i + 1 << "," << N << ","
                      << std::chrono::duration_cast<std::chrono::nanoseconds>(expr).count() << ","
                      << std::chrono::duration_cast<std::chrono::nanoseconds>(gpu_vec).count() << ","
                      << std::chrono::duration_cast<std::chrono::nanoseconds>(old_impl).count() << std::endl;
        }
    }
}  
    

int main() {
    const size_t N = 50'000'000;
    const size_t loop_count_1 = 10;
    const size_t loop_count_2 = 100;
    const size_t loop_count_3 = 1000;
    
    auto warmup_expr = twointensivecomputations_expr(N, 1);
    auto warmup_gpu_vec = twointensivecomputations_gpu_vec(N, 1);
    auto warmup_old = twointensivecomputations_old(N, 1);

    /* for(int i = 0; i < 2^31-1; i++) {
        twointensivecomputations_expr(N, 1);
        twointensivecomputations_gpu_vec(N, 1);
        twointensivecomputations_old(N, 1);
        if (i % 1000000 == 0) {
            std::cout << "Warmup iteration: " << i << " - ";
        }
    } */

    std::cout << "Warmup completed :" << warmup_expr.count() << "s, "
              << warmup_gpu_vec.count() << "s, "
              << warmup_old.count() << "s" << std::endl;


    //two_intenvisve_benchmark();
    ten_simple_computations_benchmark(10'000, 50'000'000);
                


}