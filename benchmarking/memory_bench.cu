#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <iomanip>
#include <cmath>
#include <fstream>

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

struct IntensiveComputation {
    __device__ float operator()(float x) const {
        for (int i = 0; i < 100; ++i) {
            x = sinf(x) * cosf(x) + logf(x + 1.0f);
        }
        return x;
    }
};

__global__ void computeKernel(float* data, size_t N) {
    IntensiveComputation op;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        data[idx] = op(data[idx]);
    }
}

void runWarmup(size_t N) {
    const size_t bytes = N * sizeof(float);
    float* hostPageable = (float*)malloc(bytes);
    float* hostPinned;
    hipHostMalloc((void**)&hostPinned, bytes);
    float* device;
    hipMalloc((void**)&device, bytes);
    float* managed;
    hipMallocManaged(&managed, bytes);

    for (size_t i = 0; i < N; ++i) {
        hostPageable[i] = i;
        hostPinned[i] = i;
        managed[i] = i;
    }

    for (int i = 0; i < 3; ++i) {
        hipMemcpy(device, hostPageable, bytes, hipMemcpyHostToDevice);
        hipMemcpy(hostPageable, device, bytes, hipMemcpyDeviceToHost);
        hipMemcpy(device, hostPinned, bytes, hipMemcpyHostToDevice);
        hipMemcpy(hostPinned, device, bytes, hipMemcpyDeviceToHost);
        computeKernel<<<(N + 255) / 256, 256>>>(managed, N);
        hipDeviceSynchronize();
    }

    free(hostPageable);
    hipHostFree(hostPinned);
    hipFree(device);
    hipFree(managed);
}

void benchmarkMemoryInitialization(const size_t MIN_N, const size_t MAX_N, const size_t NUMB_REPEAT) {
    std::cout << "N,Repetition,Pinned Init (ns),Pageable Init (ns),Managed Init (ns)\n";

    for (size_t N = MIN_N; N <= MAX_N; N *= 2) {
        const size_t bytes = N * sizeof(float);
        for (size_t repeat = 0; repeat < NUMB_REPEAT; ++repeat) {
            auto start = std::chrono::high_resolution_clock::now();
            float* pinned;
            hipHostMalloc((void**)&pinned, bytes);
            for (size_t i = 0; i < N; ++i) pinned[i] = i;
            auto end = std::chrono::high_resolution_clock::now();
            auto pinnedInit = end - start;
            hipHostFree(pinned);

            start = std::chrono::high_resolution_clock::now();
            float* pageable = (float*)malloc(bytes);
            for (size_t i = 0; i < N; ++i) pageable[i] = i;
            end = std::chrono::high_resolution_clock::now();
            auto pageableInit = end - start;
            free(pageable);

            start = std::chrono::high_resolution_clock::now();
            float* managed;
            hipMallocManaged(&managed, bytes);
            for (size_t i = 0; i < N; ++i) managed[i] = i;
            hipDeviceSynchronize();
            end = std::chrono::high_resolution_clock::now();
            auto managedInit = end - start;
            hipFree(managed);

            std::cout << N << "," << repeat << ","
                      << std::fixed << std::setprecision(0)
                      << pinnedInit.count() << ","
                      << pageableInit.count() << ","
                      << managedInit.count() << "\n";
        }
    }
}

void benchmarkMemoryTransfers(const size_t MIN_N, const size_t MAX_N, const size_t NUMB_REPEAT) {
    std::cout << "N,Repetition,Pinned Transfer (ns),Pageable Transfer (ns),Managed Kernel (ns),Managed Prefetch+Kernel (ns)\n";

    for (size_t N = MIN_N; N <= MAX_N; N *= 2) {
        const size_t bytes = N * sizeof(float);

        float* pinned;
        hipHostMalloc((void**)&pinned, bytes);
        float* pageable = (float*)malloc(bytes);
        float* managed;
        hipMallocManaged(&managed, bytes);
        float* device;
        hipMalloc((void**)&device, bytes);

        for (size_t i = 0; i < N; ++i) {
            pinned[i] = i;
            pageable[i] = i;
            managed[i] = i;
        }
        hipDeviceSynchronize();

        for (size_t repeat = 0; repeat < NUMB_REPEAT; ++repeat) {
            auto start = std::chrono::high_resolution_clock::now();
            hipMemcpy(device, pinned, bytes, hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            computeKernel<<<(N + 255) / 256, 256>>>(device, N);
            hipDeviceSynchronize();
            hipMemcpy(pinned, device, bytes, hipMemcpyDeviceToHost);
            auto end = std::chrono::high_resolution_clock::now();
            auto pinnedTransfer = end - start;

            start = std::chrono::high_resolution_clock::now();
            hipMemcpy(device, pageable, bytes, hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            computeKernel<<<(N + 255) / 256, 256>>>(device, N);
            hipDeviceSynchronize();
            hipMemcpy(pageable, device, bytes, hipMemcpyDeviceToHost);
            end = std::chrono::high_resolution_clock::now();
            auto pageableTransfer = end - start;

            start = std::chrono::high_resolution_clock::now();
            computeKernel<<<(N + 255) / 256, 256>>>(managed, N);
            hipDeviceSynchronize();
            end = std::chrono::high_resolution_clock::now();
            auto managedKernel = end - start;

            start = std::chrono::high_resolution_clock::now();
            hipMemPrefetchAsync(managed, bytes, 0);
            computeKernel<<<(N + 255) / 256, 256>>>(managed, N);
            hipDeviceSynchronize();
            end = std::chrono::high_resolution_clock::now();
            auto managedPrefetch = end - start;

            std::cout << N << "," << repeat << ","
                      << std::fixed << std::setprecision(0)
                      << pinnedTransfer.count() << ","
                      << pageableTransfer.count() << ","
                      << managedKernel.count() << ","
                      << managedPrefetch.count() << "\n";
        }

        hipHostFree(pinned);
        free(pageable);
        hipFree(managed);
        hipFree(device);
    }
}

void benchmarkMemoryInitAndTransfer(const size_t MIN_N, const size_t MAX_N, const size_t NUMB_REPEAT) {
    std::cout << "N,Repetition,Pinned Init+Transfer (ns),Pageable Init+Transfer (ns),Managed Init+Kernel+Prefetch (ns)\n";

    for (size_t N = MIN_N; N <= MAX_N; N *= 2) {
        const size_t bytes = N * sizeof(float);

        for (size_t repeat = 0; repeat < NUMB_REPEAT; ++repeat) {
            auto start = std::chrono::high_resolution_clock::now();
            float* pinned;
            hipHostMalloc((void**)&pinned, bytes);
            for (size_t i = 0; i < N; ++i) pinned[i] = i;
            float* device;
            hipMalloc((void**)&device, bytes);
            hipMemcpy(device, pinned, bytes, hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            computeKernel<<<(N + 255) / 256, 256>>>(device, N);
            hipDeviceSynchronize();
            hipMemcpy(pinned, device, bytes, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            auto end = std::chrono::high_resolution_clock::now();
            auto pinnedTime = end - start;
            hipHostFree(pinned);
            hipFree(device);

            start = std::chrono::high_resolution_clock::now();
            float* pageable = (float*)malloc(bytes);
            for (size_t i = 0; i < N; ++i) pageable[i] = i;
            hipMalloc((void**)&device, bytes);
            hipMemcpy(device, pageable, bytes, hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            computeKernel<<<(N + 255) / 256, 256>>>(device, N);
            hipDeviceSynchronize();
            hipMemcpy(pageable, device, bytes, hipMemcpyDeviceToHost);
            end = std::chrono::high_resolution_clock::now();
            auto pageableTime = end - start;
            free(pageable);
            hipFree(device);

            start = std::chrono::high_resolution_clock::now();
            float* managed;
            hipMallocManaged(&managed, bytes);
            for (size_t i = 0; i < N; ++i) managed[i] = i;
            hipMemPrefetchAsync(managed, bytes, 0);
            computeKernel<<<(N + 255) / 256, 256>>>(managed, N);
            hipDeviceSynchronize();
            end = std::chrono::high_resolution_clock::now();
            auto managedTime = end - start;
            hipFree(managed);

            std::cout << N << "," << repeat << ","
                      << std::fixed << std::setprecision(0)
                      << pinnedTime.count() << ","
                      << pageableTime.count() << ","
                      << managedTime.count() << "\n";
        }
    }
}

size_t estimateMaxN(size_t buffers_required = 3, double safety_factor = 0.9) {
    size_t free_mem, total_mem;
    hipMemGetInfo(&free_mem, &total_mem);
    size_t usable_bytes = static_cast<size_t>(free_mem * safety_factor);
    size_t max_floats = usable_bytes / (sizeof(float) * buffers_required);
    std::cout << "Estimated max N: " << max_floats << "\n";
    return max_floats;
}


int main() {
    const size_t MIN_N = 10'000;
    const size_t MAX_N = estimateMaxN(4);
    const size_t NUMB_REPEAT = 20;

    runWarmup(MAX_N);
    std::cout << "Running Init Benchmark\n";
    std::ofstream outFile("sheet/mem_init_fix.csv");
    auto cout_buf = std::cout.rdbuf();
    std::cout.rdbuf(outFile.rdbuf());
    benchmarkMemoryInitialization(MIN_N, MAX_N, NUMB_REPEAT);
    std::cout.rdbuf(cout_buf);
    outFile.close();
    std::cout << "Running Transfer Benchmark\n";
    std::ofstream outFile2("sheet/mem_transfer_fix.csv");
    cout_buf = std::cout.rdbuf();
    std::cout.rdbuf(outFile2.rdbuf());
    benchmarkMemoryTransfers(MIN_N, MAX_N, NUMB_REPEAT);
    std::cout.rdbuf(cout_buf);
    outFile2.close();
    std::cout << "Running Init and Transfer Benchmark\n";
    std::ofstream outFile3("sheet/mem_init_transfer_fix.csv");
    cout_buf = std::cout.rdbuf();
    std::cout.rdbuf(outFile3.rdbuf());
    benchmarkMemoryInitAndTransfer(MIN_N, MAX_N, NUMB_REPEAT);
    std::cout.rdbuf(cout_buf);
    outFile3.close();

    return 0;
}
