#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <algorithm>
#include <cmath>
#include <list>
#include <hip/hip_runtime.h>


template <typename T, typename Func>
__global__ void mapKernel(T* d_array, int size, Func func) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_array[idx] = func(d_array[idx]);
    }
}


struct IntensiveComputation {
    __device__ float operator()(float x) const {
        for (int i = 0; i < 100; ++i) { 
            x = sin(x) * cos(x) + log(x + 1.0f);
        }
        return x;
    }
};



float CpuIntensiveComputation(float x) {
    for (int i = 0; i < 100; ++i) { 
        x = sin(x) * cos(x) + log(x + 1.0f);
    }
    return x;
}



template <typename Iterator, typename Func>
void map_impl(Iterator& container, Func& func) {
    using T = typename Iterator::value_type;
    size_t size = container.size();
    T* d_array;
    size_t bytes = size * sizeof(T);
    
    hipMalloc(&d_array, bytes);
    hipMemcpy(d_array, container.data(), bytes, hipMemcpyHostToDevice);
    
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    
    mapKernel<<<numBlocks, blockSize>>>(d_array, size, func);
    hipDeviceSynchronize();
    hipMemcpy(container.data(), d_array, bytes, hipMemcpyDeviceToHost);
    hipFree(d_array);
    
}

template <typename Iterator, typename Func>
void map_impl(Iterator& container, Func& func, Iterator& output) {
    using T = typename Iterator::value_type;
    size_t size = container.size();
    T* d_array;
    size_t bytes = size * sizeof(T);
    
    hipMalloc(&d_array, bytes);
    hipMemcpy(d_array, container.data(), bytes, hipMemcpyHostToDevice);
    
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    
    mapKernel<<<numBlocks, blockSize>>>(d_array, size, func);
    hipDeviceSynchronize();
    hipMemcpy(output.data(), d_array, bytes, hipMemcpyDeviceToHost);
    hipFree(d_array);
    
}

template <typename Iterator, typename Func>
void map(Iterator& container, Func& func) {
    using T = typename Iterator::value_type;
    std::vector<T> temp; 
    if constexpr (!std::is_same_v<Iterator, std::vector<T>>){
        for (auto it = container.begin(); it != container.end(); ++it) {
            temp.push_back(*it);
        }
        map_impl(temp, func);
        std::copy(temp.begin(), temp.end(), container.begin());
    }
    else{
        map_impl(container, func);
    }
}

//Assumes input and output have the same size and type
template <typename Iterator, typename Func>
void map(Iterator& container, Func& func, Iterator& output) {
    using T = typename Iterator::value_type;
    std::vector<T> temp; 
    if constexpr (!std::is_same_v<Iterator, std::vector<T>>){
        for (auto it = container.begin(); it != container.end(); ++it) {
            temp.push_back(*it);
        }
        map_impl(temp, func);
        std::copy(temp.begin(), temp.end(), output.begin());
    }
    else{
        map_impl(container, func, output);
    }
}


template <typename Container, typename Func>
void cpuMap(Container& container, Func func) {
    for (auto& elem : container) {
        elem = func(elem);
    }
}

template <typename Container1, typename Container2>
void compareAndPrint(const std::string& name1, const Container1& container1,
                     const std::string& name2, const Container2& container2,
                     const std::string& operationName, double duration1, double duration2,
                     double tolerance = 1e-6) {

    if (container1.size() != container2.size()) {
        std::cout << "Error: Containers " << name1 << " and " << name2 << " have different sizes.\n";
        return;
    }

    bool resultsMatch = true;
    auto iter1 = container1.begin();
    auto iter2 = container2.begin();
    for (size_t i = 0; iter1 != container1.end() && iter2 != container2.end(); ++iter1, ++iter2, ++i) {
        if (std::abs(*iter1 - *iter2) > tolerance) {
            std::cout << name1 << "[" << i << "] = " << *iter1 << " "
                      << name2 << "[" << i << "] = " << *iter2 << "\n";
            resultsMatch = false;
            break;
        }
    }

    std::cout << operationName << " " << name1 << " Time: " << duration1 << " ms\n";
    std::cout << operationName << " " << name2 << " Time: " << duration2 << " ms\n";
    std::cout << operationName << " Results Match: " << (resultsMatch ? "Yes" : "No") << "\n\n";
}

int main() {
    const size_t N = 10000000;  

    //N elementos com 2.0f
    std::vector<float> cudaVec(N, 2.0f);
    std::vector<float> cpuVec(N, 2.0f);
    std::list<float> cudaVecList(N, 2.0f);

    std::vector<float> cudaVec_out(N);
    std::list<float> cudaVecList_out(N);


    auto startCuda_out = std::chrono::high_resolution_clock::now();
    map(cudaVec, IntensiveComputation(), cudaVec_out);
    auto endCuda_out = std::chrono::high_resolution_clock::now();
    auto cudaDuration_out = std::chrono::duration_cast<std::chrono::milliseconds>(endCuda_out - startCuda_out);

    auto startCudaList_out = std::chrono::high_resolution_clock::now();
    map(cudaVecList, IntensiveComputation(), cudaVecList_out);
    auto endCudaList_out = std::chrono::high_resolution_clock::now();
    auto cudaDurationList_out = std::chrono::duration_cast<std::chrono::milliseconds>(endCudaList_out - startCudaList_out);

    
    auto startCuda = std::chrono::high_resolution_clock::now();
    map(cudaVec, IntensiveComputation());
    auto endCuda = std::chrono::high_resolution_clock::now();
    auto cudaDuration = std::chrono::duration_cast<std::chrono::milliseconds>(endCuda - startCuda);

    auto startCudaList = std::chrono::high_resolution_clock::now();
    map(cudaVecList, IntensiveComputation());
    auto endCudaList = std::chrono::high_resolution_clock::now();
    auto cudaDurationList = std::chrono::duration_cast<std::chrono::milliseconds>(endCudaList - startCudaList);

    
    auto startCpu = std::chrono::high_resolution_clock::now();
    std::transform(cpuVec.begin(),cpuVec.end(),cpuVec.begin(), CpuIntensiveComputation);
    auto endCpu = std::chrono::high_resolution_clock::now();
    auto cpuDuration = std::chrono::duration_cast<std::chrono::milliseconds>(endCpu - startCpu);

    // Compare cudaVec and cpuVec
    compareAndPrint("cudaVec", cudaVec, "cpuVec", cpuVec, "Map", cudaDuration.count(), cpuDuration.count());

    // Compare cudaVecList and cpuVec
    compareAndPrint("cudaVecList", cudaVecList, "cpuVec", cpuVec, "Map List", cudaDurationList.count(), cpuDuration.count());

    // Compare cudaVecWithOutput and cpuVec
    compareAndPrint("cudaVecWithOutput", cudaVec, "cpuVec", cpuVec, "Map", cudaDuration_out.count(), cpuDuration.count());

    // Compare cudaVecListWithOutput and cpuVec
    compareAndPrint("cudaVecListWithOutput", cudaVecList, "cpuVec", cpuVec, "Map List", cudaDurationList_out.count(), cpuDuration.count());


    return 0;
}
