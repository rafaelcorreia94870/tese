
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <string>
#include <numeric>

struct DoubleIt {
    int operator()(int x) const { return 2 * x; }
};

struct AddTen {
    int operator()(int x) const { return x + 10; }
};

struct SquareIt {
    int operator()(int x) const { return x * x; }
};

template <typename T> class Vector;

template <typename BaseExpr, typename Op>
struct MapExpr {
    BaseExpr base_expr;
    Op op;

    MapExpr(BaseExpr b_expr, Op operation)
        : base_expr(b_expr), op(operation) {}

    void print() const {
        std::cout << "MapExpr: ";
        std::cout << "Base size: " << base_expr.size() << ", Operation: " << typeid(Op).name() << std::endl;
        for (size_t i = 0; i < base_expr.size(); ++i) {
            std::cout << "Index " << i << ": " << base_expr[i] << " -> " << op(base_expr[i]) << std::endl;
        }
        std::cout << "End of MapExpr" << std::endl;
    }

    auto operator[](size_t i) const {
        return op(base_expr[i]);
    }

    size_t size() const {
        return base_expr.size();
    }

    template <typename NextOp>
    auto map(NextOp next_op) const {
        return MapExpr<const MapExpr&, NextOp>(*this, next_op);
    }
};

template <typename T>
class Vector {
private:
    std::vector<T> data;

public:
    explicit Vector(size_t n) : data(n) {}
    Vector(const std::vector<T>& initial_data) : data(initial_data) {}

    T& operator[](size_t i) { return data[i]; }
    const T& operator[](size_t i) const { return data[i]; }
    size_t size() const { return data.size(); }

    std::vector<T>& get_data() { return data; }
    const std::vector<T>& get_data() const { return data; }

    void print(const std::string& label = "") const {
        if (!label.empty()) {
            std::cout << label << ": ";
        }
        for (const T& val : data) {
            std::cout << val << " ";
        }
        std::cout << std::endl;
    }

    template <typename Op>
    auto map(Op op) const {
        return MapExpr<const Vector&, Op>(*this, op);
    }

    template <typename BaseExpr, typename Op>
    Vector& operator=(const MapExpr<BaseExpr, Op>& expr) {
        if (data.size() != expr.size()) {
            data.resize(expr.size());
        }

        expr.print();

        for (size_t i = 0; i < data.size(); ++i) {
            data[i] = expr[i];
        }
        return *this;
    }

    Vector& operator=(const Vector& other) {
        if (this != &other) {
            data = other.data;
        }
        return *this;
    }
};

int main() {

    Vector<int> my_vec(5);
    std::iota(my_vec.get_data().begin(), my_vec.get_data().end(), 1);
    my_vec.print("Original Vector");

    Vector<int> result_vec(5);

    std::cout << "\n--- Example 1: my_vec.map(DoubleIt()).map(AddTen()).map(SquareIt()) ---\n";
    result_vec = my_vec.map(DoubleIt()).map(AddTen()).map(SquareIt());
    result_vec.print("Result 1");
    std::cout << "Expected: 144 196 256 324 400\n";


    std::cout << "\n--- Example 2: my_vec.map(SquareIt()).map(DoubleIt()) ---\n";
    result_vec = my_vec.map(SquareIt()).map(DoubleIt());
    result_vec.print("Result 2");
    std::cout << "Expected: 2 8 18 32 50\n";

    return 0;
}